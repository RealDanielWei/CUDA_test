
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void checkId(){
	printf("threadIdx: (%d, %d, %d)	blockIdx: (%d, %d, %d)	blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n", 
	threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
	blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z
	);
}

void cudaFunction(){
	checkId <<< 3, 3 >>> ();
}
